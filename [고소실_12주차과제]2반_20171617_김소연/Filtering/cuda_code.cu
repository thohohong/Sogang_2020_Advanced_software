#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

#define BLOCK_SIZE 32



void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2

__constant__ float constant_gaussian_kernel[ 25 ];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Gaussian_kernel_no_shared(IN unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	/*Todo*/
	const unsigned block_id = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;
	
	unsigned char mean;
	int w = 2;
	int w0 = 5;

	for (int c = 0; c < width; c++) {
		mean = 0;
		for (int l = w * -1; l <= w; l++) {
			for (int k = w * -1; k <= w; k++) {
				if (id + k >= height || id + k < 0 || c + l >= width || c + l < 0) mean += 0;
				else mean += d_bitmaps[(id + k) * width + c + l] * constant_gaussian_kernel[(w + k) * w0 + w + l];
			}
		}
		d_Gaussian[id * width + c] = mean;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	/*Todo*/
	const unsigned block_id = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;
	
	int w0 = 5;
	int w = 2;

	
	if (thread_id == 0) {
		for (int i = 1; i <= w; i++) {
			if (id - i < 0) {
				for (int j = 0; j < width; j++) {
					sharedBuffer[(w - i) * width + j] = 0;
				}
			}
			else if (id - i >= 0 && id - i <= height) {
				for (int j = 0; j < width; j++) {
					sharedBuffer[(w - i) * width + j] = d_bitmaps[(id - i) * width + j];
				}
			}

		}
	}
	
	
	if (thread_id == BLOCK_SIZE - 1) {
		for (int i = 1; i <= w; i++) {
			if (id + i >= height) {
				for (int j = 0; j < width; j++) {
					sharedBuffer[(BLOCK_SIZE + w + i - 1) * width + j] = 0;
				}
			}
			else if (id + i < height) {
				for (int j = 0; j < width; j++) {
					sharedBuffer[(BLOCK_SIZE + w + i - 1) * width + j] = d_bitmaps[(id + i) * width + j];
				}
			}
			
		}
	}
	
	
	for (int i = 0; i < width; i++) {
		sharedBuffer[(thread_id + w) * width + i] = d_bitmaps[id * width + i];
	}
	//complete buffer

	__syncthreads();

	unsigned char mean;
	
	for (int c = 0; c < width; c++) {
		mean = 0;
		for (int l = w * -1; l <= w; l++) {
			for (int k = w * -1; k <= w; k++) {
				//if (id + k >= height || id + k < 0 || c + l >= width || c + l < 0) mean += 0;
				if (c + l >= width || c + l < 0) mean += 0;
				else mean += sharedBuffer[(thread_id + w + k) * width + c + l] * constant_gaussian_kernel[(w + k) * w0 + w + l];
			}
		}
		d_Gaussian[id * width + c] = mean;
	}

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel(){
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float *p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9]= _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] =p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL( constant_gaussian_kernel), p_gaussian_kernel, sizeof( float ) * 25 );

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char *p_bitmaps, OUT unsigned char *p_Gaussian, long width, long height, int Shared_flag)
{
	/*Todo*/
	int device_time;
	hipError_t cudaStatus;

	CUDA_CALL(hipSetDevice(0));

	Set_Gaussain_Kernel();

	unsigned char* d_bitmaps, * d_Gaussian;
	size_t size = sizeof(unsigned char) * width * height;
	CUDA_CALL(hipMalloc(&d_bitmaps, size));
	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, size, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_Gaussian, size));

	dim3 dimBlock(BLOCK_SIZE, 1);
	dim3 dimGrid(height / dimBlock.x, 1);

	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag) {
	case NO_SHARED :
		Gaussian_kernel_no_shared << < dimGrid, dimBlock >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED :
		Gaussian_kernel_shared << < dimGrid, dimBlock, sizeof(unsigned char) * (BLOCK_SIZE + 2 * 2) * width >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	}

	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipGetLastError());
	CUDA_CALL(hipDeviceSynchronize());
	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, size, hipMemcpyDeviceToHost));

	hipFree(d_bitmaps);
	hipFree(d_Gaussian);

	return device_time;
}