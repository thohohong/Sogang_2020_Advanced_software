#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 128
#define DATA_SIZE (1 << 26)
#define N_EQUATIONS 1<<20

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
float* A;
float* B;
float* C;

float* X0_CPU;
float* X1_CPU;
float* FX0_CPU;
float* FX1_CPU;

float* X0_GPU;
float* X1_GPU;
float* FX0_GPU;
float* FX1_GPU;


hipError_t find_roots_GPU(float* A, float* B, float* C, float* X0, float* X1, float* FX0, float* FX1, int n);

__global__ void find_roots_Kernel(float* A, float* B, float* C, float* X0, float* X1, float* FX0, float* FX1)
{
    /*Thread Idx*/
    int row;
    int col;
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    /*Todo*/

    float a, b, c, d, x0, x1, tmp;
    a = A[id]; b = B[id]; c = C[id];
    d = sqrtf(b * b - 4.0f * a * c);
    tmp = 1.0f / (2.0f * a);
    X0[id] = x0 = (-b - d) * tmp;
    X1[id] = x1 = (-b + d) * tmp;
    FX0[id] = (a * x0 + b) * x0 + c;
    FX1[id] = (a * x1 + b) * x1 + c;

}


void find_roots_CPU(float* A, float* B, float* C, float* X0, float* X1, float* FX0, float* FX1, int n) {
    float a, b, c, d, x0, x1, tmp;

    for (int i = 0; i < n; i++) {
        a = A[i]; b = B[i]; c = C[i];
        d = sqrtf(b * b - 4.0f * a * c);
        tmp = 1.0f / (2.0f * a);
        X0[i] = x0 = (-b - d) * tmp;
        X1[i] = x1 = (-b + d) * tmp;
        FX0[i] = (a * x0 + b) * x0 + c;
        FX1[i] = (a * x1 + b) * x1 + c;
    }
}

void init_bin_file(int size) {
    srand((unsigned)time(NULL));
    printf("***Binary File init Start!!\n");
    FILE* fp_A = fopen("A.bin", "wb");
    FILE* fp_B = fopen("B.bin", "wb");
    FILE* fp_C = fopen("C.bin", "wb");

    float random_num;
    for (int i = 0; i < size; i++) {
        random_num = (float)rand() / RAND_MAX;
        fwrite(&random_num, sizeof(float), 1, fp_A);
    }
    for (int i = 0; i < size; i++) {
        random_num = (float)rand() / RAND_MAX;
        fwrite(&random_num, sizeof(float), 1, fp_B);
    }
    for (int i = 0; i < size; i++) {
        random_num = (float)rand() / RAND_MAX;
        fwrite(&random_num, sizeof(float), 1, fp_C);
    }

    fclose(fp_A);
    fclose(fp_B);
    fclose(fp_C);
    printf("***Binary File init End!!\n\n");
}

void read_bin_file()
{
    printf("***Binary File Read Start!!\n");
    FILE* fp_A = fopen("A.bin", "rb");
    FILE* fp_B = fopen("B.bin", "rb");
    FILE* fp_C = fopen("C.bin", "rb");

    A = (float*)malloc(sizeof(float) * N_EQUATIONS);
    B = (float*)malloc(sizeof(float) * N_EQUATIONS);
    C = (float*)malloc(sizeof(float) * N_EQUATIONS);

    X0_CPU = (float*)malloc(sizeof(float) * N_EQUATIONS);
    X1_CPU = (float*)malloc(sizeof(float) * N_EQUATIONS);
    FX0_CPU = (float*)malloc(sizeof(float) * N_EQUATIONS);
    FX1_CPU = (float*)malloc(sizeof(float) * N_EQUATIONS);

    X0_GPU = (float*)malloc(sizeof(float) * N_EQUATIONS);
    X1_GPU = (float*)malloc(sizeof(float) * N_EQUATIONS);
    FX0_GPU = (float*)malloc(sizeof(float) * N_EQUATIONS);
    FX1_GPU = (float*)malloc(sizeof(float) * N_EQUATIONS);

    fread(A, sizeof(float), N_EQUATIONS, fp_A);
    fread(B, sizeof(float), N_EQUATIONS, fp_B);
    fread(C, sizeof(float), N_EQUATIONS, fp_C);

    fclose(fp_A);
    fclose(fp_B);
    fclose(fp_C);

    printf("***Binary File Read End!!\n\n");
}

int main()
{
    init_bin_file(N_EQUATIONS);
    read_bin_file();

    printf("The problem size is %d.\n", N_EQUATIONS);
    int i;
    FILE* fp_X0, * fp_X1, * fp_FX0, * fp_FX1;

    //CPU 
    printf("***find_roots_CPU Start!!\n");
    CHECK_TIME_START;
    find_roots_CPU(A, B, C, X0_CPU, X1_CPU, FX0_CPU, FX1_CPU, N_EQUATIONS);
    CHECK_TIME_END(compute_time);
    printf("***find_roots_CPU End!!\n\n");

    printf("CPU time = %.6f\n", compute_time);
    //GPU

    printf("***find_roots_GPU Start!!\n");
    find_roots_GPU(A, B, C, X0_GPU, X1_GPU, FX0_GPU, FX1_GPU, N_EQUATIONS);
    printf("***find_roots_GPU End!!\n\n");

    printf("GPU time = %.6f\n", device_time);


    for (i = 0; i < N_EQUATIONS; i++)
    {
        if (X0_CPU[i] != X0_GPU[i])
        {
            printf("%u != %u\n", X0_CPU[i], X0_GPU[i]);
            break;
        }
    }


    if (i == N_EQUATIONS)
        printf("***Kernel execution Success!!\n\n");

    // Write the output array into the output file.
    fp_X0 = fopen("X0.bin", "wb");
    fp_X1 = fopen("X1.bin", "wb");
    fp_FX0 = fopen("FX0.bin", "wb");
    fp_FX1 = fopen("FX1.bin", "wb");

    for (i = 0; i < N; i++) {
        fwrite(&X0_GPU[i], sizeof(float), 1, fp_X0);
        fwrite(&X1_GPU[i], sizeof(float), 1, fp_X1);
        fwrite(&FX0_GPU[i], sizeof(float), 1, fp_FX0);
        fwrite(&FX1_GPU[i], sizeof(float), 1, fp_FX1);
    }

    fclose(fp_X0);
    fclose(fp_X1);
    fclose(fp_FX0);
    fclose(fp_FX1);

    printf("end!!\n\n");

    return 0;
}


hipError_t find_roots_GPU(float* A, float* B, float* C, float* X0, float* X1, float* FX0, float* FX1, int n)
{


    hipError_t cudaStatus;
    /*Todo*/

    cudaStatus = hipSetDevice(0);


    float* d_A, * d_B, * d_C, * d_X0, * d_X1, * d_FX0, * d_FX1;

    size_t size = sizeof(float) * n;
    CUDA_CALL(hipMalloc(&d_A, size))
        CUDA_CALL(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));

    CUDA_CALL(hipMalloc(&d_B, size))
        CUDA_CALL(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

    CUDA_CALL(hipMalloc(&d_C, size))
        CUDA_CALL(hipMemcpy(d_C, C, size, hipMemcpyHostToDevice));

    CUDA_CALL(hipMalloc(&d_X0, size))
        CUDA_CALL(hipMalloc(&d_X1, size))
        CUDA_CALL(hipMalloc(&d_FX0, size))
        CUDA_CALL(hipMalloc(&d_FX1, size))
        dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(n / dimBlock.x, 1);

    CHECK_TIME_INIT_GPU()
        CHECK_TIME_START_GPU()
        find_roots_Kernel << < dimGrid, dimBlock >> > (d_A, d_B, d_C, d_X0, d_X1, d_FX0, d_FX1);
    CHECK_TIME_END_GPU(device_time)
        CHECK_TIME_DEST_GPU();

    CUDA_CALL(hipGetLastError())
        CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(X0, d_X0, size, hipMemcpyDeviceToHost))
        CUDA_CALL(hipMemcpy(X1, d_X1, size, hipMemcpyDeviceToHost))
        CUDA_CALL(hipMemcpy(FX0, d_FX0, size, hipMemcpyDeviceToHost))
        CUDA_CALL(hipMemcpy(FX1, d_FX1, size, hipMemcpyDeviceToHost))

        Error:

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X0);
    hipFree(d_X1);
    hipFree(d_FX0);
    hipFree(d_FX1);



    return cudaStatus;
}